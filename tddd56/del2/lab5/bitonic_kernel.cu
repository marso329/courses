#include "hip/hip_runtime.h"

// This is not really C++-code but pretty plain C code, but we compile it
// as C++ so we can integrate with CUDA seamlessly.

// If you plan on submitting your solution for the Parallel Sorting Contest,
// please keep the split into main file and kernel file, so we can easily
// insert other data.
#include <stdio.h>

#define MAX_SHARED 12288

__device__ 
static void exchange(int *i, int *j)
{
	int k;
	k = *i;
	*i = *j;
	*j = k;
}



__global__
void bitonic_kernel(int *data, int k, int j,int N)
{
	
//index
int i = threadIdx.x + blockIdx.x*blockDim.x;
	int ixj=i^j; // Calculate indexing!
if ( k<blockDim.x && j<blockDim.x){
		__shared__ int shared_data[MAX_SHARED];
		shared_data[i%blockDim.x]=data[i];
		__syncthreads();
		bool change=false;
        if ((ixj)>i)
        {  	
		if ((i&k)==0 && shared_data[i%blockDim.x]>shared_data[ixj%blockDim.x]){
		 exchange(&shared_data[i%blockDim.x],&shared_data[ixj%blockDim.x]);
		 change=true;
		 }
		if ((i&k)!=0 && shared_data[i%blockDim.x]<shared_data[ixj%blockDim.x]){
		 exchange(&shared_data[i%blockDim.x],&shared_data[ixj%blockDim.x]);
        change=true;
        }
        } 
        if (change){
 data[i]=shared_data[i%blockDim.x];
 data[ixj]=shared_data[ixj%blockDim.x];
}

}
else{

        if ((ixj)>i)
        {  
		if ((i&k)==0 && data[i]>data[ixj]) exchange(&data[i],&data[ixj]);
		if ((i&k)!=0 && data[i]<data[ixj]) exchange(&data[i],&data[ixj]);
        } 

}

}



// No, this is not GPU code yet but just a copy of the CPU code, but this
// is where I want to see your GPU code!
void bitonic_gpu(int *data, int N)
{
/**
 int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("shared memory:%i\n", prop.sharedMemPerBlock);
  }
  printf("sizeof int: %i",sizeof(int));
**/



  int *dev_data;
  int size = N * sizeof(int);


  hipMalloc((void**)&dev_data, size);
  hipMemcpy(dev_data, data, size, hipMemcpyHostToDevice);
//                _________________________________________________   
//block is linear |_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|_|
//								1,0----------k,0
  dim3 dimBlock (min(N ,1024), 1);
  dim3 dimGrid (N / 1024  + 1, 1);

int j,k;
  for (k=2;k<=N;k=2*k) // Outer loop, double size for each step
  {
    for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
    {
  	bitonic_kernel<<<dimGrid, dimBlock>>>(dev_data, k, j,N);
  	hipDeviceSynchronize();
    }
  }


  hipMemcpy(data, dev_data, size, hipMemcpyDeviceToHost);

  hipFree(dev_data);

}
